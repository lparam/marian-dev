#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kernels/dropout.h"
#include "kernels/tensor_operators.h"

#define CUDA_CALL(x)                                  \
  do {                                                \
    if((x) != hipSuccess) {                          \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      exit(1);                                        \
    }                                                 \
  } while(0)

#define CURAND_CALL(x)                                \
  do {                                                \
    if((x) != HIPRAND_STATUS_SUCCESS) {                \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      exit(1);                                        \
    }                                                 \
  } while(0)

namespace marian {

__global__ void gScale(float* data, int n, float p) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  while(index < n) {
    data[index] = (data[index] < p) / p;
    index += gridDim.x * blockDim.x;
  }
}

void Dropout(Tensor tensor, float p, hiprandGenerator_t gen) {
  int n = tensor->size();
  CURAND_CALL(hiprandGenerateUniform(gen, tensor->data(), n));

  int numThreads = std::min(n, 512);
  int numBlocks = n / numThreads + (n % numThreads != 0);

  gScale<<<numBlocks, numThreads>>>(tensor->data(), n, 1.f - p);
}

void Gaussian(Tensor tensor, float mean, float stddev, hiprandGenerator_t gen) {
  int n = tensor->size();
  // @TODO: fix misalignment with new allocator and use true gaussian noise
  CURAND_CALL(hiprandGenerateNormal(gen, tensor->data(), n, mean, stddev));
  //CURAND_CALL(hiprandGenerateUniform(gen, tensor->data(), n));
  //Element(_1 = 2.f * stddev * _1 - stddev, tensor);
}

}
