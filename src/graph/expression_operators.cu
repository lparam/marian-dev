#include "hip/hip_runtime.h"
#include "graph/expression_operators.h"
#include "kernels/sparse.h"

#include "graph/node_operators.h"
#include "graph/node_operators_binary.h"
#include "graph/node_operators_unary.h"

namespace marian {

Expr debug(Expr a, const std::string& message) {
  a->debug(message);
  return a;
}

Expr rows(Expr a, const std::vector<size_t>& indeces) {
  return Expression<RowsNodeOp>(a, indeces);
}

Expr cols(Expr a, const std::vector<size_t>& indeces) {
  return Expression<ColsNodeOp>(a, indeces);
}

Expr logit(Expr a) {
  return Expression<LogitNodeOp>(a);
}

Expr relu(Expr a) {
  return Expression<ReLUNodeOp>(a);
}

Expr log(Expr a) {
  return Expression<LogNodeOp>(a);
};

Expr exp(Expr a) {
  return Expression<ExpNodeOp>(a);
};

Expr operator-(Expr a) {
  return Expression<NegNodeOp>(a);
};

Expr softmax(Expr a, Expr mask) {
  return Expression<SoftmaxNodeOp>(a, mask);
}

Expr logsoftmax(Expr a) {
  return Expression<LogSoftmaxNodeOp>(a);
}

/*********************************************************/

Expr operator+(Expr a, Expr b) {
  return Expression<PlusNodeOp>(a, b);
}

Expr operator-(Expr a, Expr b) {
  return Expression<MinusNodeOp>(a, b);
}

Expr operator*(Expr a, Expr b) {
  return Expression<MultNodeOp>(a, b);
}

Expr operator/(Expr a, Expr b) {
  return Expression<DivNodeOp>(a, b);
}

/*********************************************************/

Expr operator+(Expr a, float b) {
  return Expression<ScalarAddNodeOp>(a, b);
}

Expr operator+(float a, Expr b) {
  return Expression<ScalarAddNodeOp>(b, a);
}

Expr operator-(Expr a, float b) {
  return Expression<ScalarAddNodeOp>(a, -b);
}

Expr operator-(float a, Expr b) {
  return Expression<ScalarAddNodeOp>(-b, a);
}

Expr operator*(float a, Expr b) {
  return Expression<ScalarMultNodeOp>(b, a);
}

Expr operator*(Expr a, float b) {
  return Expression<ScalarMultNodeOp>(a, b);
}

Expr operator/(Expr a, float b) {
  return Expression<ScalarMultNodeOp>(a, 1.f / b);
}

/*********************************************************/

Expr concatenate(const std::vector<Expr>& concats, keywords::axis_k ax) {
  return Expression<ConcatenateNodeOp>(concats, ax);
}

Expr reshape(Expr a, Shape shape) {
  return Expression<ReshapeNodeOp>(a, shape);
}

Expr flatten(Expr a) {
  Shape shape = {a->shape().elements()};
  return Expression<ReshapeNodeOp>(a, shape);
}

Expr sum(Expr a, keywords::axis_k ax) {
  return Expression<SumNodeOp>(a, ax);
}

Expr mean(Expr a, keywords::axis_k ax) {
  return Expression<MeanNodeOp>(a, ax);
}

Expr scalar_product(Expr a, Expr b, keywords::axis_k ax) {
  return Expression<ScalarProductNodeOp>(a, b, ax);
}

Expr weighted_average(Expr in, Expr weights, keywords::axis_k ax) {
  auto p = scalar_product(in, weights, ax);
  auto s = sum(weights, ax);
  return p / s;
}

Expr dot(Expr a, Expr b) {
  return Expression<DotNodeOp>(a, b);
}

Expr transpose(Expr a) {
  return Expression<TransposeNodeOp>(a);
}

Expr step(Expr a, size_t step) {
  return Expression<TimestepNodeOp>(a, step);
}

Expr cross_entropy(Expr a, Expr b) {
  auto sOrig = a->shape();
  auto sOut = a->shape();
  Shape sTemp({sOrig[0] * sOrig[2] * sOrig[3], sOrig[1], 1, 1});
  sOut.set(1, 1);
  return reshape(Expression<CrossEntropyNodeOp>(reshape(a, sTemp), b), sOut);
}

Expr affine(Expr a, Expr b, Expr c) {
  std::vector<Expr> nodes = {a, b, c};
  return Expression<AffineNodeOp>(nodes);
}

Expr plus(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr tanh(const std::vector<Expr>& nodes) {
  return Expression<TanhNodeOp>(nodes);
}

Expr logit(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr relu(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr sqrt(Expr a, float eps) {
  return Expression<SqrtNodeOp>(a, eps);
}

Expr square(Expr a) {
  return Expression<SquareNodeOp>(a);
}

Expr layer_norm(Expr x, Expr gamma, Expr beta) {
  std::vector<Expr> nodes = {x, gamma};
  if(beta)
    nodes.push_back(beta);
  return Expression<LayerNormalizationOp>(nodes);
}

// Expr batch_norm(Expr x, Expr gamma, Expr beta) {
//  auto mju = mean(x, keywords::axis=0);
//  auto xmmju = x - mju;
//  auto std = sqrt(mean(square(xmmju), keywords::axis=0), 1e-9);
//
//  if(beta)
//    return gamma * (xmmju / std) + beta;
//  else
//    return gamma * (xmmju / std);
//}

Expr shift(Expr a, Shape shift) {
  return Expression<ShiftNodeOp>(a, shift);
}

Expr lexical_bias(Expr logits, Expr att, float eps, Ptr<sparse::CSR> lf) {
  return Expression<LexicalProbNodeOp>(logits, att, eps, lf);
}

#ifdef CUDNN

Expr convolution(Expr x,
  Expr filters, Expr bias,
  int padHeight, int padWidth,
  int strideHeight, int strideWidth)
{
  std::vector<Expr> nodes = {x, filters, bias};
  return Expression<ConvolutionOp>(nodes,
      padHeight, padWidth, strideHeight, strideWidth);
}

Expr avg_pooling(
    Expr x,
    int height, int width,
    int padHeight, int padWidth,
    int strideHeight, int strideWidth)
{
  return Expression<PoolingOp>(x,
      height, width,
      padHeight, padWidth,
      strideHeight, strideWidth,
      PoolingOp::Mode::AVERAGE_POOLING);
}

Expr max_pooling(
    Expr x,
    int height, int width,
    int padHeight, int padWidth,
    int strideHeight, int strideWidth)
{
  return Expression<PoolingOp>(x,
      height, width,
      padHeight, padWidth,
      strideHeight, strideWidth,
      PoolingOp::Mode::MAX_POOLING);
}

#endif

}
