#include "hip/hip_runtime.h"
#include "rnn/attention.h"

#include "graph/node_operators_binary.h"
#include "kernels/tensor_operators.h"

namespace marian {

namespace rnn {

struct AttentionNodeOp : public NaryNodeOp {
  AttentionNodeOp(const std::vector<Expr>& nodes)
      : NaryNodeOp(nodes, keywords::shape = newShape(nodes)) {}

  Shape newShape(const std::vector<Expr>& nodes) {
    Shape shape = nodes[1]->shape();

    Shape vaShape = nodes[0]->shape();
    Shape keysShape = nodes[1]->shape();
    Shape queryShape = nodes[2]->shape();

    for(int i = 0; i < 2; ++i) {
      UTIL_THROW_IF2(keysShape[i] != queryShape[i]
                     && keysShape[i] != 1
                     && queryShape[i] != 1,
                     "Shapes cannot be broadcasted");
      shape.set(i, std::max(keysShape[i], queryShape[i]));
    }

    UTIL_THROW_IF2(vaShape[0] != shape[1] || vaShape[1] != 1, "Wrong size");

    shape.set(1, 1);
    shape.set(2, keysShape[2]);
    shape.set(3, queryShape[2] * queryShape[3]);

    return shape;
  }

  NodeOps forwardOps() {
    return {NodeOp(Att(val_,
                       child(0)->val(),
                       child(1)->val(),
                       child(2)->val()))};
  }

  NodeOps backwardOps() {
    return {
      NodeOp(AttBack(child(0)->grad(),
                     child(1)->grad(),
                     child(2)->grad(),
                     child(0)->val(),
                     child(1)->val(),
                     child(2)->val(),
                     adj_);)
    };
  }

  // do not check if node is trainable
  virtual void runBackward(const NodeOps& ops) {
    for(auto&& op : ops)
      op();
  }

  const std::string type() { return "Att-ops"; }

  const std::string color() { return "yellow"; }
};

Expr attOps(Expr va, Expr keys, Expr query) {
  std::vector<Expr> nodes{va, keys, query};

  int dimBatch = keys->shape()[0];
  int dimWords = keys->shape()[2];
  int dimBeam = query->shape()[2] * query->shape()[3];
  return reshape(Expression<AttentionNodeOp>(nodes),
                 {dimWords, dimBatch, 1, dimBeam});
}

}
}
