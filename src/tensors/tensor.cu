#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "tensors/tensor.h"
#include "kernels/tensor_operators.h"
#include "kernels/cuda_helpers.h"

namespace marian {

__global__ void gFill(float* d_in, int size, float val) {
  for(int bid = 0; bid < size; bid += blockDim.x * gridDim.x) {
    int index = bid + threadIdx.x + blockDim.x * blockIdx.x;
    if (index < size) {
      d_in[index] = val;
    }
  }
}

float TensorBase::get(size_t i) {
   hipSetDevice(device_);
   float temp;
   CUDA_CHECK(hipMemcpy(&temp, data_ + i, sizeof(float),
              hipMemcpyDeviceToHost));
   hipStreamSynchronize(0);
   return temp;
 }

void TensorBase::set(size_t i, float value) {
  hipSetDevice(device_);
  CUDA_CHECK(hipMemcpy(data_ + i, &value, sizeof(float),
             hipMemcpyHostToDevice));
  hipStreamSynchronize(0);
}

void TensorBase::get(std::vector<float> &v) {
  CUDA_CHECK(hipSetDevice(device_));
  v.resize(size());
  CUDA_CHECK(hipMemcpy(v.data(), data_, size() * sizeof(float),
             hipMemcpyDeviceToHost));
  hipStreamSynchronize(0);
}

void TensorBase::set(float value) {
  hipSetDevice(device_);
  int threads = std::min(512, (int)size());
  int blocks = (size() / threads) + (size() % threads != 0);
  gFill<<<blocks, threads>>>(data_, size(), value);
  hipStreamSynchronize(0);
}

void TensorBase::set(const std::vector<float> &v) {
  CUDA_CHECK(hipSetDevice(device_));
  CUDA_CHECK(hipMemcpy(data_, v.data(), v.size() * sizeof(float),
             hipMemcpyHostToDevice));
  hipStreamSynchronize(0);
}

void TensorBase::setSparse(const std::vector<size_t> &k,
                           const std::vector<float> &v) {
  hipSetDevice(device_);
  SetSparse(data_, k, v);
  hipStreamSynchronize(0);
}


void TensorBase::copyFrom(Tensor in) {
    hipSetDevice(device_);
    CUDA_CHECK(hipMemcpy(data_ , in->data() , in->size() * sizeof(float),
                          hipMemcpyDefault));
    hipStreamSynchronize(0);
}

std::string TensorBase::debug() {
  hipSetDevice(device_);
  std::stringstream strm;
  assert(shape_.size());
  strm << "shape=" << shape_[0];
  for(int i = 1; i < shape_.size(); ++i)
     strm << "x" << shape_[i];
  strm << " size=" << shape_.elements()
     << " (" << shape_.elements() * sizeof(float) << "B)";
  strm << " device=" << device_ << std::endl;

  // values
  size_t totSize = shape_.elements();
  std::vector<Float> values(totSize);
  get(values);


  size_t dispCols = 5;
  strm << std::fixed << std::setprecision(8) << std::setfill(' ');
  for(size_t l = 0; l < shape()[3]; ++l) {
    for(size_t k = 0; k < shape()[2]; ++k) {
       strm << "[ ";
       if(shape()[0] > 10) {
          for (size_t i = 0; i < shape()[0] && i < dispCols; ++i) {
             if(i > 0)
               strm << std::endl << "  ";
             for (size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
             if(shape()[1] > dispCols)
                strm << "... ";
             for (size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
          }
          strm << std::endl << "  ...";
          for (size_t i = shape()[0] - dispCols; i < shape()[0]; ++i) {
             if(i > 0)
               strm << std::endl << "  ";
             for (size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
             if(shape()[1] > dispCols)
                strm << "... ";
             for (size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
          }
       }
       else {
          for (size_t i = 0; i < shape()[0] && i < 10; ++i) {
             if(i > 0)
               strm << std::endl << "  ";
             for (size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
             if(shape()[1] > dispCols)
                strm << "... ";
             for (size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
               strm << std::setw(12)
                    << values[  i * shape().stride(0)
                              + j * shape().stride(1)
                              + k * shape().stride(2)
                              + l * shape().stride(3) ] << " ";
             }
          }
       }
       strm << "]" << std::endl;
    }
  }
  return strm.str();
}

DeviceGPU::~DeviceGPU() {
  hipSetDevice(device_);
  if(data_) {
    CUDA_CHECK(hipFree(data_));
  }
  hipDeviceSynchronize();
}

void DeviceGPU::reserve(size_t size) {
   hipSetDevice(device_);

   UTIL_THROW_IF2(size < size_, "New size must be larger than old size");

   if(data_) {
     // Allocate memory by going through host memory
     float *temp = new float[size_];
     CUDA_CHECK(hipMemcpy(temp, data_, size_* sizeof(float),
                hipMemcpyDeviceToHost));
     CUDA_CHECK(hipFree(data_));
     CUDA_CHECK(hipMalloc(&data_, size * sizeof(float)));
     CUDA_CHECK(hipMemcpy(data_, temp, size_* sizeof(float),
                hipMemcpyHostToDevice));
     delete[] temp;
   }
   else {
      CUDA_CHECK(hipMalloc(&data_, size * sizeof(float)));
   }

   size_ = size;
}

Tensor operator<<(Tensor t, const std::vector<float>& v) {
  t->set(v);
  return t;
}

Tensor operator>>(Tensor t, std::vector<float>& v) {
  t->get(v);
  return t;
}

}
