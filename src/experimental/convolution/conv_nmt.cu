#include "hip/hip_runtime.h"
#include "models/amun.h"
#include "models/conv_nmt.h"
#include "models/encdec.h"

namespace marian {

ConvolutionalEncoderState::ConvolutionalEncoderState(
    Expr attContext, Expr srcContext, Expr mask, Ptr<data::CorpusBatch> batch)
    : attContext_(attContext),
      srcContext_(srcContext),
      mask_(mask),
      batch_(batch) {}

const std::vector<size_t>& ConvolutionalEncoderState::getSourceWords() {
  return batch_->front()->indeces();
}

Expr ConvolutionalEncoderState::getSrcContext() {
  return srcContext_;
}

Expr ConvolutionalEncoderState::getContext() {
  return attContext_;
}

Expr ConvolutionalEncoderState::getMask() {
  return mask_;
}

Ptr<EncoderState> ConvolutionalEncoder::build(Ptr<ExpressionGraph> graph,
                                              Ptr<data::CorpusBatch> batch,
                                              size_t batchIdx) {
  using namespace keywords;

  int dimSrcVoc = options_->get<std::vector<int>>("dim-vocabs")[batchIdx];
  int dimSrcEmb = options_->get<int>("dim-emb");
  int maxSrcLength = options_->get<int>("max-length");

  float dropoutSrc = inference_ ? 0 : options_->get<float>("dropout-src");

  auto xEmb = Embedding("Wemb", dimSrcVoc, dimSrcEmb)(graph);
  auto posEmb = Embedding("Wemb_pos", maxSrcLength, dimSrcEmb)(graph);

  Expr x, xMask;

  std::tie(x, xMask) = prepareSource(xEmb, posEmb, batch, batchIdx);

  std::string convType = options_->get<std::string>("conv-enc-type");

  if(dropoutSrc) {
    int srcWords = x->shape()[2];
    auto srcWordDrop = graph->dropout(dropoutSrc, {1, 1, srcWords});
    x = dropout(x, mask = srcWordDrop);
  }

  Expr attContext;
  Expr srcContext;
  if(convType == "pooling") {
    attContext = Pooling("enc_pooling")(x, xMask);
    srcContext = x;
  } else if(convType == "full") {
    attContext = Convolution("conv_att", 3, 1, 1)(x, xMask);
    srcContext = Convolution("conv_src", 3, 1, 1)(x, xMask);
  } else {
    LOG("Unknown type of convolutional encoder");
  }

  return New<ConvolutionalEncoderState>(attContext, srcContext, xMask, batch);
}

std::tuple<Expr, Expr> ConvolutionalEncoder::prepareSource(
    Expr emb, Expr posEmb, Ptr<data::CorpusBatch> batch, size_t index) {
  using namespace keywords;

  auto& wordIndeces = batch->at(index)->indeces();

  auto& mask = batch->at(index)->mask();

  std::vector<size_t> posIndeces;

  for(size_t iPos = 0; iPos < batch->at(index)->batchWidth(); ++iPos) {
    for(size_t i = 0; i < batch->at(index)->batchSize(); ++i) {
      if(iPos < posEmb->shape()[0]) {
        posIndeces.push_back(iPos);
      } else {
        posIndeces.push_back(posEmb->shape()[0] - 1);
      }
    }
  }

  int batchSize = batch->size();
  int dimEmb = emb->shape()[1];
  int batchLength = batch->at(index)->batchWidth();

  auto graph = emb->graph();

  auto xWord
      = reshape(rows(emb, wordIndeces), {batchSize, dimEmb, batchLength});
  auto xPos
      = reshape(rows(posEmb, posIndeces), {batchSize, dimEmb, batchLength});
  auto x = xWord + xPos;
  auto xMask = graph->constant(shape = {batchSize, 1, batchLength},
                               init = inits::from_vector(mask));
  return std::make_tuple(x, xMask);
}

Ptr<DecoderState> ConvolutionalDecoder::startState(Ptr<EncoderState> encState) {
  using namespace keywords;

  auto meanContext = weighted_average(
      std::static_pointer_cast<ConvolutionalEncoderState>(encState)
          ->getSrcContext(),
      encState->getMask(),
      axis = 2);

  bool layerNorm = options_->get<bool>("layer-normalization");
  auto start = Dense("ff_state",
                     options_->get<int>("dim-rnn"),
                     activation = act::tanh,
                     normalize = layerNorm)(meanContext);
  std::vector<Expr> startStates(options_->get<size_t>("layers-dec"), start);
  return New<DecoderStateAmun>(start, nullptr, encState);
}

Ptr<DecoderState> ConvolutionalDecoder::step(Ptr<ExpressionGraph> graph,
                                             Ptr<DecoderState> state) {
  using namespace keywords;

  int dimTrgVoc = options_->get<std::vector<int>>("dim-vocabs").back();
  int dimTrgEmb = options_->get<int>("dim-emb");
  int dimDecState = options_->get<int>("dim-rnn");
  bool layerNorm = options_->get<bool>("layer-normalization");
  bool skipDepth = options_->get<bool>("skip");
  size_t decoderLayers = options_->get<size_t>("layers-dec");

  float dropoutRnn = inference_ ? 0 : options_->get<float>("dropout-rnn");
  float dropoutTrg = inference_ ? 0 : options_->get<float>("dropout-trg");

  auto stateAmun = std::dynamic_pointer_cast<DecoderStateAmun>(state);
  auto embeddings = stateAmun->getTargetEmbeddings();

  if(dropoutTrg) {
    int trgWords = embeddings->shape()[2];
    auto trgWordDrop = graph->dropout(dropoutTrg, {1, 1, trgWords});
    embeddings = dropout(embeddings, mask = trgWordDrop);
  }

  if(!attention_) {
    attention_ = New<GlobalAttention>(
        "decoder",
        state->getEncoderState(),
        std::static_pointer_cast<ConvolutionalEncoderState>(
            state->getEncoderState())
            ->getSrcContext(),
        dimDecState,
        dropout_prob = dropoutRnn,
        normalize = layerNorm);
  }

  if(!rnn)
    rnn = New<RNN<CGRU>>(graph,
                         "decoder",
                         dimTrgEmb,
                         dimDecState,
                         attention_,
                         dropout_prob = dropoutRnn,
                         normalize = layerNorm);
  auto stateOut = (*rnn)(embeddings, stateAmun->getState());

  bool single = stateAmun->doSingleStep();

  auto alignedContextsVec = attention_->getContexts();
  auto alignedContext = single ?
                            alignedContextsVec.back() :
                            concatenate(alignedContextsVec, keywords::axis = 2);

  //// 2-layer feedforward network for outputs and cost
  auto logitsL1 = Dense(
      "ff_logit_l1", dimTrgEmb, activation = act::tanh, normalize = layerNorm)(
      embeddings, stateOut, alignedContext);

  auto logitsOut = Dense("ff_logit_l2", dimTrgVoc)(logitsL1);

  return New<DecoderStateAmun>(stateOut, logitsOut, state->getEncoderState());
}

}  // namespace marian
