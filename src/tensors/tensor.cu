#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "kernels/cuda_helpers.h"
#include "kernels/tensor_operators.h"
#include "tensors/tensor.h"

namespace marian {

__global__ void gFill(float *d_in, int size, float val) {
  for(int bid = 0; bid < size; bid += blockDim.x * gridDim.x) {
    int index = bid + threadIdx.x + blockDim.x * blockIdx.x;
    if(index < size) {
      d_in[index] = val;
    }
  }
}

float TensorBase::get(size_t i) {
  hipSetDevice(device_);
  float temp;
  CUDA_CHECK(
      hipMemcpy(&temp, data() + i, sizeof(float), hipMemcpyDeviceToHost));
  hipStreamSynchronize(0);
  return temp;
}

void TensorBase::set(size_t i, float value) {
  hipSetDevice(device_);
  CUDA_CHECK(
      hipMemcpy(data() + i, &value, sizeof(float), hipMemcpyHostToDevice));
  hipStreamSynchronize(0);
}

void TensorBase::get(std::vector<float> &v) {
  CUDA_CHECK(hipSetDevice(device_));
  v.resize(size());
  CUDA_CHECK(hipMemcpy(
      v.data(), data(), size() * sizeof(float), hipMemcpyDeviceToHost));
  hipStreamSynchronize(0);
}

void TensorBase::set(float value) {
  hipSetDevice(device_);
  int threads = std::min(512, (int)size());
  int blocks = (size() / threads) + (size() % threads != 0);
  gFill<<<blocks, threads>>>(data(), size(), value);
  hipStreamSynchronize(0);
}

void TensorBase::set(const std::vector<float> &v) {
  CUDA_CHECK(hipSetDevice(device_));
  CUDA_CHECK(hipMemcpy(
      data(), v.data(), v.size() * sizeof(float), hipMemcpyHostToDevice));
  hipStreamSynchronize(0);
}

void TensorBase::setSparse(const std::vector<size_t> &k,
                           const std::vector<float> &v) {
  hipSetDevice(device_);
  SetSparse(data(), k, v);
  hipStreamSynchronize(0);
}

void TensorBase::copyFrom(Tensor in) {
  hipSetDevice(device_);
  CUDA_CHECK(hipMemcpy(
      data(), (float*)in->data(), in->size() * sizeof(float), hipMemcpyDefault));
  hipStreamSynchronize(0);
}

std::string TensorBase::debug() {
  hipSetDevice(device_);
  std::stringstream strm;
  assert(shape_.size());
  strm << shape_;
  strm << " device=" << device_;
  strm << " ptr=" << (size_t)memory_->data();
  strm << " bytes=" << memory_->size();
  strm << std::endl;

  // values
  size_t totSize = shape_.elements();
  std::vector<float> values(totSize);
  get(values);

  size_t dispCols = 5;
  strm << std::fixed << std::setprecision(8) << std::setfill(' ');
  for(size_t l = 0; l < shape()[3]; ++l) {
    for(size_t k = 0; k < shape()[2]; ++k) {
      strm << "[ ";
      if(shape()[0] > 10) {
        for(size_t i = 0; i < shape()[0] && i < dispCols; ++i) {
          if(i > 0)
            strm << std::endl << "  ";
          for(size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
          if(shape()[1] > dispCols)
            strm << "... ";
          for(size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
        }
        strm << std::endl << "  ...";
        for(size_t i = shape()[0] - dispCols; i < shape()[0]; ++i) {
          if(i > 0)
            strm << std::endl << "  ";
          for(size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
          if(shape()[1] > dispCols)
            strm << "... ";
          for(size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
        }
      } else {
        for(size_t i = 0; i < shape()[0] && i < 10; ++i) {
          if(i > 0)
            strm << std::endl << "  ";
          for(size_t j = 0; j < shape()[1] && j < dispCols; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
          if(shape()[1] > dispCols)
            strm << "... ";
          for(size_t j = shape()[1] - dispCols; j < shape()[1]; ++j) {
            strm << std::setw(12)
                 << values[i * shape().stride(0) + j * shape().stride(1)
                           + k * shape().stride(2)
                           + l * shape().stride(3)]
                 << " ";
          }
        }
      }
      strm << "]" << std::endl;
    }
  }
  return strm.str();
}

Tensor operator<<(Tensor t, const std::vector<float> &v) {
  t->set(v);
  return t;
}

Tensor operator>>(Tensor t, std::vector<float> &v) {
  t->get(v);
  return t;
}
}
