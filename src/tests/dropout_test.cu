#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include <boost/chrono.hpp>
#include <boost/timer/timer.hpp>
#include <vector>

#include "common/config.h"
#include "layers/param_initializers.h"
#include "marian.h"

using namespace marian;
using namespace keywords;

int main(int argc, char** argv) {
  auto c = New<Config>(argc, argv);

  auto g = New<ExpressionGraph>();
  g->setDevice(0);
  g->reserveWorkspaceMB(512);

  for(int i = 0; i < 10; ++i) {
    g->clear();
    auto mask = g->dropout(0.2, {10, 3072});
    debug(mask, "mask");
    g->forward();
  }

  return 0;
}
