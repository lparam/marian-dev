#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include "marian.h"
#include "layers/highway.h"
#include "layers/convolution.h"

using namespace marian;

bool test_vectors(const std::vector<float>& output, const std::vector<float>& corrent) {
  if (output.size() != corrent.size()) {
    return false;
  }

  for (size_t i = 0; i < output.size(); ++i) {
    if (fabsf(output[i] - corrent[i]) > 0.0001f) {
      return false;
    }
  }
  return true;
}

std::vector<float> strideMask(const std::vector<float>& mask, int batchSize, int stride) {
  std::vector<float> strided;

  for (size_t wordIdx = 0; wordIdx < mask.size(); wordIdx += stride * batchSize) {
    for (size_t j = wordIdx; j < wordIdx + batchSize; ++j) {
      strided.push_back(mask[j]);
    }
  }
  return strided;
}

int main(int argc, char** argv) {
  auto graph = New<ExpressionGraph>();
  graph->setDevice(1);
  graph->reserveWorkspaceMB(128);

  int dimBatch = 2;
  int dimWord = 2;
  int batchLength = 7;
  int numLayers = 1;

  int elemNum = dimBatch * dimWord * batchLength * numLayers;

  std::vector<float> embData(elemNum);
  std::vector<float> embMask(dimBatch * batchLength);

  for (size_t i = 0; i < embData.size() ; ++i) {
    // embData[2 * dimWord * (i / dimWord) + (i % dimWord)] = float(1);// / (i + 1.0f);
    // embData[2 * dimWord * (i / dimWord) + (i % dimWord) + dimWord] = float(1); // / (i + 1.0f);
    embData[i] = float(i); // / (i + 1.0f);
  }

  for (auto& v : embMask) {
    v = 1.0f;
  }
  embMask.back() = 0.0f;

  auto x = graph->param("x", {dimBatch, dimWord, batchLength},
                        keywords::init=inits::from_vector(embData));


  auto xMask = graph->constant({dimBatch, 1, batchLength},
                               keywords::init=inits::from_vector(embMask));

  std::vector<int> convWidths({1, 2});
  std::vector<int> convSizes({1, 1});

  auto convolution = MultiConvolution("multi_conv", dimWord, convWidths, convSizes)(x, xMask);

  auto highway = Highway("highway", 4)(convolution);
  auto idx = graph->constant({120, 1}, keywords::init=inits::zeros);
  auto ce = cross_entropy(highway, idx);
  auto cost = mean(sum(ce, keywords::axis=2), keywords::axis=0);


  debug(x, "x");
  debug(cost, "COST");
  debug(convolution, "CONVOLUTION");
  debug(highway, "highway");
  debug(ce, "ce");

  graph->forward();
  graph->backward();

  std::vector<float> output;
  std::vector<float> output2;

  for (auto v : output2) std::cerr << v << " ";
  std::cerr << std::endl;

  return 0;
}
