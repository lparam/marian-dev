#include "hip/hip_runtime.h"

#include <thrust/transform_reduce.h>

#include "kernels/cuda_helpers.h"
#include "kernels/tensor_operators.h"

#include "3rd_party/reduce_all.h"

namespace marian {

#define CUDA_FLT_MAX 1.70141e+38

struct isnan_test {
  __host__ __device__ bool operator()(const float a) const {
      return isnan(a);
  }
};

bool IsNan(Tensor in) {
  thrust::device_ptr<float> begin = thrust::device_pointer_cast(in->data());
  thrust::device_ptr<float> end = thrust::device_pointer_cast(in->data() + in->size());
  return thrust::transform_reduce(begin, end, isnan_test(), 0, thrust::plus<bool>());
}

__global__ void gConcatN(float* out,
                         ShapeGPU outShape,
                         float** ins,
                         ShapeGPU inShape,
                         size_t* lengths,
                         size_t num,
                         size_t axis) {

  int dims[4];
  int length = outShape.elements();

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      int offset = 0;
      size_t i = 0;
      const float* in = ins[0];
      size_t inLength = lengths[0];

      outShape.dims(index, dims);

      while(dims[axis] >= offset + lengths[i] && i < num) {
        offset += lengths[i];
        inLength = lengths[i + 1];
        in = ins[i + 1];
        i++;
      }

      inShape.set(axis, inLength);
      dims[axis] -= offset;
      int inIndex = inShape.bindex(dims);

      out[index] = in[inIndex];
    }
  }
}

void ConcatN(Tensor out, const std::vector<Tensor>& ins, int axis) {
  int length = out->size();

  size_t num = ins.size();

  std::vector<float*> vins;
  std::vector<size_t> vlengths;
  for(auto in : ins) {
    vins.push_back(in->data());
    vlengths.push_back(in->shape()[axis]);
  }

  float** d_ins;
  CUDA_CHECK(hipMalloc(&d_ins, num * sizeof(float*)));
  CUDA_CHECK(hipMemcpy(d_ins,
                        vins.data(),
                        num * sizeof(float*),
                        hipMemcpyHostToDevice));

  size_t* d_lengths;
  CUDA_CHECK(hipMalloc(&d_lengths, num * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_lengths,
                        vlengths.data(),
                        num * sizeof(size_t),
                        hipMemcpyHostToDevice));

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  auto inShape = ins[0]->shape();
  inShape.set(axis, 1);

  gConcatN<<<blocks, threads>>>(out->data(),
                                out->shape(),
                                d_ins,
                                inShape,
                                d_lengths,
                                num,
                                axis);

  CUDA_CHECK(hipFree(d_ins));
  CUDA_CHECK(hipFree(d_lengths));
}

__global__ void gSplitN(float* in,
                        ShapeGPU inShape,
                        float** outs,
                        ShapeGPU outShape,
                        size_t* lengths,
                        size_t num,
                        size_t axis) {

  int dims[4];
  int length = inShape.elements();

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      int offset = 0;
      size_t i = 0;
      float* out = outs[0];
      size_t outLength = lengths[0];

      inShape.dims(index, dims);

      while(dims[axis] >= offset + lengths[i] && i < num) {
        offset += lengths[i];
        outLength = lengths[i + 1];
        out = outs[i + 1];
        i++;
      }

      outShape.set(axis, outLength);
      dims[axis] -= offset;
      int outIndex = outShape.bindex(dims);

      out[outIndex] = in[index];
    }
  }
}

void SplitN(std::vector<Tensor>& outs, Tensor in, int axis) {
  int length = in->size();

  size_t num = outs.size();

  std::vector<float*> vouts;
  std::vector<size_t> vlengths;
  for(auto out : outs) {
    vouts.push_back(out->data());
    vlengths.push_back(out->shape()[axis]);
  }

  float** d_outs;
  CUDA_CHECK(hipMalloc(&d_outs, num * sizeof(float*)));
  CUDA_CHECK(hipMemcpy(d_outs,
                        vouts.data(),
                        num * sizeof(float*),
                        hipMemcpyHostToDevice));

  size_t* d_lengths;
  CUDA_CHECK(hipMalloc(&d_lengths, num * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_lengths,
                        vlengths.data(),
                        num * sizeof(size_t),
                        hipMemcpyHostToDevice));

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  auto outShape = outs[0]->shape();
  outShape.set(axis, 1);

  gSplitN<<<blocks, threads>>>(in->data(),
                               in->shape(),
                               d_outs,
                               outShape,
                               d_lengths,
                               num,
                               axis);

  CUDA_CHECK(hipFree(d_outs));
  CUDA_CHECK(hipFree(d_lengths));
}

__global__ void gTranspose4D(float* out,
                             ShapeGPU outShape,
                             const float* in,
                             const ShapeGPU inShape,
                             const int permute[4]) {

  int length = outShape.elements();
  int dims1[4];
  int dims2[4];

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      outShape.dims(index, dims1);

      for(int i = 0; i < 4; ++i)
        dims2[i] = dims1[permute[i]];

      int inIndex = inShape.bindex(dims2);

      out[index] = in[inIndex];
    }
  }
}

void Transpose4D(Tensor out, Tensor in, const std::array<int, 4>& permute) {
  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  int* d_permute;
  CUDA_CHECK(hipMalloc(&d_permute, 4 * sizeof(int)));
  CUDA_CHECK(hipMemcpy(d_permute,
                        permute.data(),
                        4 * sizeof(int),
                        hipMemcpyHostToDevice));

  gTranspose4D<<<blocks, threads>>>(out->data(),
                                    out->shape(),
                                    in->data(),
                                    in->shape(),
                                    d_permute);

  CUDA_CHECK(hipFree(d_permute));
}

__global__ void gSoftmax(float* out,
                         ShapeGPU outShape,
                         const float* in,
                         const float* mask,
                         const ShapeGPU maskShape) {

  int rows = outShape[0] * outShape[2] * outShape[3];
  int cols = outShape[1];

  bool broadcast = outShape != maskShape;
  int dims[4];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = -CUDA_FLT_MAX;  // mask
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {

          float mVal = 1.f;
          if(mask) {
            int mIndex = id + j * cols;
            if(broadcast) {
              outShape.dims(mIndex, dims);
              mIndex = maskShape.bindex(dims);
            }
            mVal = mask[mIndex];
          }

          if(mVal && sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {

          float mVal = 1.f;
          if(mask) {
            int mIndex = id + j * cols;
            if(broadcast) {
              outShape.dims(mIndex, dims);
              mIndex = maskShape.bindex(dims);
            }
            mVal = mask[mIndex];
          }

          float ex = 0;
          if(mVal)
            ex = __expf(sp[id] - max);
          so[id] = ex;

          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          so[id] = so[id] / _sum[0];
        }
      }
    }
  }
}

void Softmax(Tensor out, Tensor in, Tensor mask) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape()[0] * out->shape()[2] * out->shape()[3];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  if(mask)
    gSoftmax<<<blocks, threads, shared>>>(
        out->data(), out->shape(), in->data(), mask->data(), mask->shape());
  else
    gSoftmax<<<blocks, threads, shared>>>(
        out->data(), out->shape(), in->data(), 0, out->shape());
  // hipStreamSynchronize(0);
}

__global__ void gLogSoftmax(float* out,
                            const ShapeGPU outShape,
                            const float* in) {
  int rows = outShape[0] * outShape[2] * outShape[3];
  int cols = outShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sm = sp[id] - max;
          float ex = __expf(sm);
          so[id] = sm;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          so[id] -= __logf(_sum[0]);
      }
    }
  }
}

void LogSoftmax(Tensor out, Tensor in) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape()[0] * out->shape()[2] * out->shape()[3];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gLogSoftmax<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data());
}

///////////////////////////////////////////////////////

__global__ void gSoftmaxGrad(float* grad,
                             const float* adj,
                             const float* val,
                             const int rows,
                             const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += valRow[id] * adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float val = valRow[id] * (adjRow[id] - _sum[0]);
          if(val)
            gradRow[id] += val;
        }
      }
    }
  }
}

void SoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDevice());
  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape()[0] * grad->shape()[2] * grad->shape()[3];
  int k = grad->shape()[1];

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

__global__ void gLogSoftmaxGrad(float* grad,
                                const float* adj,
                                const float* val,
                                const int rows,
                                const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols)
          gradRow[id] += adjRow[id] - (expf(valRow[id]) * _sum[0]);
      }
    }
  }
}

void LogSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  hipSetDevice(adj->getDevice());

  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape()[0] * grad->shape()[2] * grad->shape()[3];
  int k = grad->shape()[1];

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gLogSoftmaxGrad<<<blocks, threads, shared>>>(
      grad->data(), adj->data(), val->data(), m, k);
}

///////////////////////////////////////////////////////
__global__ void gArgmax(float* out,
                        const float* data,
                        size_t rows,
                        size_t cols) {
  size_t row = blockIdx.x;
  size_t startInd = row * cols;
  float maxScore = -99999;
  size_t maxInd;
  for(size_t col = 0; col < cols; ++col) {
    size_t ind = startInd + col;
    float score = data[ind];
    if(score > maxScore) {
      maxScore = score;
      maxInd = col;
    }
  }
  out[row] = maxInd;
}

///////////////////////////////////////////////////////

void Prod(hipblasHandle_t handle,
          Tensor C,
          const Tensor A,
          const Tensor B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  hipSetDevice(C->getDevice());
  float alpha = scalar;

  size_t m = A->shape()[0] * A->shape()[2] * A->shape()[3];
  size_t k = A->shape()[1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[0];
  size_t n = B->shape()[1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[1];
  size_t ldb = B->shape()[1];
  size_t ldc = B->shape()[1];

  if(transB)
    ldc = B->shape()[0];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  hipblasSgemm(handle,
              opB,
              opA,
              n,
              m,
              k,
              &alpha,
              B->data(),
              ldb,
              A->data(),
              lda,
              &beta,
              C->data(),
              ldc);
}

void ProdBatched(
          hipblasHandle_t handle,
          Tensor C,
          const Tensor A,
          const Tensor B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  hipSetDevice(C->getDevice());
  float alpha = scalar;

  size_t batchA = A->shape()[2] * A->shape()[3];
  size_t batchB = B->shape()[2] * B->shape()[3];

  size_t m = A->shape()[0];
  size_t k = A->shape()[1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[0];
  size_t n = B->shape()[1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[1];
  size_t ldb = B->shape()[1];
  size_t ldc = B->shape()[1];

  if(transB)
    ldc = B->shape()[0];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  hipblasSgemmStridedBatched(
              handle,
              opB,
              opA,
              n,
              m,
              k,
              &alpha,
              B->data(),
              ldb,
              batchB == 1 ? 0 : n * k,
              A->data(),
              lda,
              batchA == 1 ? 0 : m * k,
              &beta,
              C->data(),
              ldc,
              n * m,
              std::max(batchA, batchB));
}

__global__ void gCopyRows(float* out,
                          const float* in,
                          size_t cols,
                          const size_t* sourceRowIdx,
                          size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = j;
      size_t srcId = sourceRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

void CopyRows(Tensor out, const Tensor in, const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t cols = in->shape()[1];
  size_t rowsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, rowsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        rowsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gCopyRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, d_indices, rowsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}

__global__ void gPasteRows(float* out,
                           const float* in,
                           size_t cols,
                           const size_t* targetRowIdx,
                           size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = targetRowIdx[j];
      size_t srcId = j;

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          atomicAdd(rowOut + i, rowIn[i]);
      }
    }
  }
}

void PasteRows(Tensor out,
               const Tensor in,
               const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t cols = in->shape()[1];
  size_t rowsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  // @TODO: turn into tensor
  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, rowsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        rowsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gPasteRows<<<blocks, threads>>>(
      out->data(), in->data(), cols, d_indices, rowsToCopy);
  CUDA_CHECK(hipFree(d_indices));
}

/////////////

__global__ void gCopyCols(float* out,
                          const float* in,
                          size_t rows,
                          size_t colsIn,
                          const size_t* sourceColIdx,
                          size_t colsOut) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsOut; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsOut)
          rowOut[i] = rowIn[sourceColIdx[i]];
      }
    }
  }
}

void CopyCols(Tensor out, const Tensor in, const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t rows = in->shape()[0] * in->shape()[2] * in->shape()[3];
  size_t cols = in->shape()[1];
  size_t colsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, colsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        colsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gCopyCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, d_indices, colsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}

__global__ void gPasteCols(float* out,
                           const float* in,
                           size_t rows,
                           size_t colsOut,
                           const size_t* targetColIdx,
                           size_t colsIn) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* rowIn = in + j * colsIn;
      float* rowOut = out + j * colsOut;

      for(int tid = 0; tid < colsIn; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsIn)
          rowOut[targetColIdx[i]] = rowIn[i];
      }
    }
  }
}

void PasteCols(Tensor out,
               const Tensor in,
               const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  size_t rows = out->shape()[0] * out->shape()[2] * out->shape()[3];
  size_t cols = out->shape()[1];
  size_t colsToCopy = indices.size();

  int threads = std::min(MAX_THREADS, (int)colsToCopy);
  int blocks = std::min(MAX_BLOCKS, (int)rows);

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, colsToCopy * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        colsToCopy * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gPasteCols<<<blocks, threads>>>(
      out->data(), in->data(), rows, cols, d_indices, colsToCopy);

  CUDA_CHECK(hipFree(d_indices));
}


__global__ void gSelect(float* out,
                        ShapeGPU outShape,
                        const float* in,
                        const ShapeGPU inShape,
                        int axis,
                        size_t* d_indices) {

  int length = outShape.elements();
  int dims[4];

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      outShape.dims(index, dims);
      dims[axis] = d_indices[dims[axis]];
      int inIndex = inShape.bindex(dims);
      out[index] = in[inIndex];
    }
  }
}

__global__ void gInsert(float* out,
                        ShapeGPU outShape,
                        const float* in,
                        const ShapeGPU inShape,
                        int axis,
                        size_t* d_indices) {

  int length = inShape.elements();
  int dims[4];

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      inShape.dims(index, dims);
      dims[axis] = d_indices[dims[index]];
      int outIndex = outShape.bindex(dims);
      out[outIndex] = in[index];
    }
  }
}

void Select(Tensor out, const Tensor in, int axis, const std::vector<size_t>& indices) {
  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, indices.size() * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        indices.size() * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gSelect<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axis,
                               d_indices);

  CUDA_CHECK(hipFree(d_indices));
}

void Insert(Tensor out, const Tensor in, int axis, const std::vector<size_t>& indices) {
  hipSetDevice(in->getDevice());

  int length = in->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, indices.size() * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        indices.size() * sizeof(size_t),
                        hipMemcpyHostToDevice));

  gInsert<<<blocks, threads>>>(out->data(),
                               out->shape(),
                               in->data(),
                               in->shape(),
                               axis,
                               d_indices);

  CUDA_CHECK(hipFree(d_indices));
}

//////////////

void Transpose(hipblasHandle_t cublasHandle, Tensor out, const Tensor in) {
  hipSetDevice(out->getDevice());
  size_t steps = in->shape()[2] * in->shape()[3];
  for(int i = 0; i < steps; i++) {
    size_t m = in->shape()[0];
    size_t n = in->shape()[1];
    float alpha = 1.0;
    float beta = 0.0;

    size_t offset = i * m * n;

    hipblasSgeam(cublasHandle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_T,
                m,
                n,
                &alpha,
                in->data() + offset,
                n,
                &beta,
                in->data() + offset,
                n,
                out->data() + offset,
                m);
  }
}

void Concatenate0(Tensor out, const std::vector<Tensor>& inputs) {
  hipSetDevice(out->getDevice());

  size_t offset = 0;
  for(auto in : inputs) {
    UTIL_THROW_IF2(out->shape()[1] != in->shape()[1],
                   "Second dimension must be equal");
    hipMemcpy(out->data() + offset,
               in->data(),
               in->size() * sizeof(float),
               hipMemcpyDeviceToDevice);
    offset += in->size();
  }
}

__global__ void gInsertCols(float* out,
                            const float* in,
                            size_t rows,
                            size_t cols,
                            size_t cols_out,
                            size_t cols_in,
                            size_t offset_out,
                            size_t offset_in) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols_out + offset_out;
      const float* rowIn = in + j * cols_in + offset_in;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

__global__ void gConcatenateAx1(float* out,
                                size_t rows,
                                const float* in1,
                                const float* in2,
                                size_t colsIn1,
                                size_t colsIn2) {
  size_t cols = colsIn1 + colsIn2;
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;
      const float* rowIn1 = in1 + j * colsIn1;
      const float* rowIn2 = in2 + j * colsIn2;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsIn1)
          rowOut[i] = rowIn1[i];
        else if(i >= colsIn1 && i < colsIn1 + colsIn2)
          rowOut[i] = rowIn2[i - colsIn1];
      }
    }
  }
}

__global__ void gConcatenateAx1(float* out,
                                size_t rows,
                                const float* in1,
                                const float* in2,
                                const float* in3,
                                size_t colsIn1,
                                size_t colsIn2,
                                size_t colsIn3) {
  size_t cols = colsIn1 + colsIn2 + colsIn3;
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;
      const float* rowIn1 = in1 + j * colsIn1;
      const float* rowIn2 = in2 + j * colsIn2;
      const float* rowIn3 = in3 + j * colsIn3;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < colsIn1)
          rowOut[i] = rowIn1[i];
        else if(i >= colsIn1 && i < colsIn1 + colsIn2)
          rowOut[i] = rowIn2[i - colsIn1];
        else if(i >= colsIn1 + colsIn2 && i < colsIn1 + colsIn2 + colsIn3)
          rowOut[i] = rowIn3[i - colsIn1 - colsIn2];
      }
    }
  }
}

void Concatenate1(Tensor out, const std::vector<Tensor>& inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape()[0] * out->shape()[2] * out->shape()[3];
  if(inputs.size() == 2) {
    int cols = out->shape()[1];
    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols);
    gConcatenateAx1<<<blocks, threads>>>(out->data(),
                                         rows,
                                         inputs[0]->data(),
                                         inputs[1]->data(),
                                         inputs[0]->shape()[1],
                                         inputs[1]->shape()[1]);
    return;
  }
  if(inputs.size() == 3) {
    int cols = out->shape()[1];
    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols);
    gConcatenateAx1<<<blocks, threads>>>(out->data(),
                                         rows,
                                         inputs[0]->data(),
                                         inputs[1]->data(),
                                         inputs[2]->data(),
                                         inputs[0]->shape()[1],
                                         inputs[1]->shape()[1],
                                         inputs[2]->shape()[1]);
    return;
  }

  size_t offset = 0;
  int cols_out = out->shape()[1];

  for(auto in : inputs) {
    UTIL_THROW_IF2(out->shape()[0] != in->shape()[0],
                   "First dimension must be equal");
    int cols_in = in->shape()[1];

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_in);

    gInsertCols<<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_in, cols_out, cols_in, offset, 0);
    offset += cols_in;
  }
}

void Concatenate(Tensor out, const std::vector<Tensor>& inputs, int ax) {
  if(ax == 1)
    Concatenate1(out, inputs);
  else
    Concatenate0(out, inputs);
}

void Deconcatenate0(std::vector<Tensor>& outputs, const Tensor in) {
  hipSetDevice(in->getDevice());

  size_t offset = 0;
  for(auto out : outputs) {
    hipMemcpy(out->data(),
               in->data() + offset,
               out->size() * sizeof(float),
               hipMemcpyDeviceToDevice);
    offset += out->size();
  }
}

void Deconcatenate1(std::vector<Tensor>& outputs, const Tensor in) {
  hipSetDevice(in->getDevice());

  size_t offset = 0;
  int rows = in->shape()[0] * in->shape()[2] * in->shape()[3];
  int cols_in = in->shape()[1];
  for(auto out : outputs) {
    UTIL_THROW_IF2(out->shape()[0] != in->shape()[0],
                   "First dimension must be equal");
    int cols_out = out->shape()[1];

    int blocks = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_out);

    gInsertCols<<<blocks, threads>>>(
        out->data(), in->data(), rows, cols_out, cols_out, cols_in, 0, offset);
    offset += cols_out;
  }
}

void Deconcatenate(std::vector<Tensor>& outputs, const Tensor in, int ax) {
  if(ax == 1)
    Deconcatenate1(outputs, in);
  else
    Deconcatenate0(outputs, in);
}

__global__ void gGRUFastForward(float* out,
                                const float* state,
                                const float* xW,
                                const float* sU,
                                const float* b,
                                const float* mask,
                                size_t rows,
                                size_t cols,
                                bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];
      float* rowOut = out + j * cols;
      const float* rowState = state + j * cols;

      const float* xWrow = xW + j * cols * 3;
      const float* sUrow = sU + j * cols * 3;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float ev1 = expf(-(xWrow[i] + sUrow[i] + b[i]));
          float r = 1.0f / (1.0f + ev1);

          int k = i + cols;
          float ev2 = expf(-(xWrow[k] + sUrow[k] + b[k]));
          float z = 1.0f / (1.0f + ev2);

          int l = i + 2 * cols;
          float h;
          if(final)
            h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
          else
            h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

          float out = (1.0f - z) * h + z * rowState[i];
          rowOut[i] = m * out + (1 - m) * rowState[i];
        }
      }
    }
  }
}

void GRUFastForward(Tensor out, std::vector<Tensor> inputs, bool final) {
  hipSetDevice(out->getDevice());

  int rows = out->shape()[0] * out->shape()[2] * out->shape()[3];
  int cols = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols,
      final);
}

__global__ void gGRUFastBackward(float* outState,
                                 float* outXW,
                                 float* outSU,
                                 float* outB,
                                 const float* state,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 const float* adj,
                                 size_t rows,
                                 size_t cols,
                                 bool final) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutState = outState + j * cols;
      float* rowOutXW = outXW + j * cols * 3;
      float* rowOutSU = outSU + j * cols * 3;

      const float* rowState = state + j * cols;
      const float* rowXW = xW + j * cols * 3;
      const float* rowSU = sU + j * cols * 3;
      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + cols;
          int l = i + 2 * cols;

          float ev1 = expf(-(rowXW[i] + rowSU[i] + b[i]));
          float r = 1.0f / (1.0f + ev1);

          float ev2 = expf(-(rowXW[k] + rowSU[k] + b[k]));
          float z = 1.0f / (1.0f + ev2);

          float h;
          if(final)
            h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
          else
            h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

          float adj = rowAdj[i];

          float t = (1 - z) * (1 - h * h);

          // df/ds
          if(outState)
            rowOutState[i] += (m * z - m + 1) * adj;

          // df/d(xW_r) ...
          float dfdxW_r = m * r * (1 - r) * t * adj;
          if(final)
            dfdxW_r *= rowSU[l] + b[l];
          else
            dfdxW_r *= rowSU[l];
          if(outXW)
            rowOutXW[i] += dfdxW_r;
          if(outSU)
            rowOutSU[i] += dfdxW_r;
          if(outB)
            atomicAdd(outB + i, dfdxW_r);

          // df/d(xW_z) ...
          float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
          if(outXW)
            rowOutXW[k] += dfdxW_z;
          if(outSU)
            rowOutSU[k] += dfdxW_z;
          if(outB)
            atomicAdd(outB + k, dfdxW_z);

          // df/d(xW_x) ...
          float dfdxW_x = m * t * adj;
          if(outXW)
            rowOutXW[l] += dfdxW_x;
          if(outSU)
            rowOutSU[l] += dfdxW_x * r;
          if(outB)
            if(final)
              atomicAdd(outB + l, dfdxW_x * r);
            else
              atomicAdd(outB + l, dfdxW_x);
        }
      }
    }
  }
}

void GRUFastBackward(std::vector<Tensor> outputs,
                     std::vector<Tensor> inputs,
                     Tensor adj,
                     bool final) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape()[0] * adj->shape()[2] * adj->shape()[3];
  int cols = adj->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols,
      final);
}

__global__ void gCrossEntropyPick(float* out,
                                  const ShapeGPU outShape,
                                  const float* in,
                                  const ShapeGPU inShape,
                                  const float* pick) {
  int rows = inShape[0];
  int cols = inShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += __expf(sp[id] - max);
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id == (int)pick[j]) {
          out[j] = __logf(_sum[0]) - sp[id] + max;
        }
      }
    }
  }
}

void CrossEntropyPick(Tensor out, Tensor in, Tensor pick) {
  hipSetDevice(out->getDevice());

  size_t m = in->shape()[0] * in->shape()[2] * in->shape()[3];
  size_t k = in->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPick<<<blocks, threads, shared>>>(
      out->data(), out->shape(), in->data(), in->shape(), pick->data());
}

__global__ void gCrossEntropyPickBackward(float* out,
                                          const ShapeGPU outShape,
                                          const float* adj,
                                          const float* in,
                                          const float* pick) {
  int rows = outShape[0];
  int cols = outShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;
      float* so = out + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          if(sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          if(_max[threadIdx.x + skip] > _max[threadIdx.x]) {
            _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = __expf(sp[id] - max);
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sub = (float)(id == (int)pick[j]);
          so[id] += adj[j] * (__expf(sp[id] - max) / _sum[0] - sub);
        }
      }
    }
  }
}

void CrossEntropyPickBackward(Tensor out, Tensor adj, Tensor a, Tensor pick) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape()[0] * out->shape()[2] * out->shape()[3];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPickBackward<<<blocks, threads, shared>>>(
      out->data(), out->shape(), adj->data(), a->data(), pick->data());
}

float L2Norm(Tensor in) {
  hipSetDevice(in->getDevice());

  uint8_t* data;
  hipMalloc(&data, sizeof(float));
  Tensor out(new TensorBase(New<MemoryPiece>(data, sizeof(float)), {1, 1}, in->getDevice()));
  ReduceAll(_1 * _1, out, in);
  float dataCpu = sqrtf(out->get(0));
  out.reset();
  hipFree(data);
  return dataCpu;
}

__global__ void gAtt(float* out,
                     const float* va,
                     const float* ctx,
                     const float* state,
                     const float* cov,
                     int m,  // total rows (batch x time x beam)
                     int k,  // depth
                     int b,  // batch size
                     int t   // time of ctx
                     ) {
  int rows = m;
  int cols = k;
  for(int bid = 0; bid < m; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* vaRow = va;
      const float* ctxRow = ctx + (j % (b * t)) * cols;
      const float* stateRow = state + (j / (b * t) + j % b) * cols;
      const float* covRow = cov ? cov + (j % (b * t)) * cols : nullptr;

      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = ctxRow[id] + stateRow[id];
          if(cov)
            z += covRow[id];
          float ex = tanhf(z) * vaRow[id];
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      out[j] = _sum[0];
    }
  }
}

void Att(Tensor out, Tensor va, Tensor context, Tensor state, Tensor coverage) {
  hipSetDevice(out->getDevice());

  size_t m = out->shape()[0] * out->shape()[2] * out->shape()[3];

  size_t b = context->shape()[0];
  size_t k = context->shape()[1];
  size_t t = context->shape()[2];

  int blocks = std::min(MAX_BLOCKS, (int)m);
  int threads = std::min(MAX_THREADS, (int)k);
  int shared = sizeof(float) * threads * 2;

  gAtt<<<blocks, threads, shared>>>(out->data(),
                                    va->data(),
                                    context->data(),
                                    state->data(),
                                    coverage ? coverage->data() : nullptr,
                                    m,
                                    k,
                                    b,
                                    t);
}

__global__ void gAttBack(float* gVa,
                         float* gContext,
                         float* gState,
                         float* gCoverage,
                         const float* va,
                         const float* context,
                         const float* state,
                         const float* coverage,
                         const float* adj,
                         int m,  // rows
                         int k,  // cols
                         int n   // batch size
                         ) {
  int rows = m;
  int cols = k;
  for(int bid = 0; bid < m; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* gcRow = gContext + j * cols;
      float* gsRow = gState + (j % n) * cols;
      float* gcovRow = gCoverage ? gCoverage + j * cols : nullptr;

      const float* cRow = context + j * cols;
      const float* sRow = state + (j % n) * cols;
      const float* covRow = coverage ? coverage + j * cols : nullptr;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float z = cRow[id] + sRow[id];
          if(coverage)
            z += covRow[id];

          float t = tanhf(z);
          float r = va[id] * (1.f - t * t);

          gcRow[id] += r * adj[j];
          gsRow[id] += r * adj[j];
          if(gCoverage)
            gcovRow[id] += r * adj[j];
          atomicAdd(gVa + id, t * adj[j]);
        }
      }
    }
  }
}

void AttBack(Tensor gVa,
             Tensor gContext,
             Tensor gState,
             Tensor gCoverage,
             Tensor va,
             Tensor context,
             Tensor state,
             Tensor coverage,
             Tensor adj) {
  hipSetDevice(adj->getDevice());

  size_t m = context->shape()[0] * context->shape()[2] * context->shape()[3];
  size_t k = context->shape()[1];

  size_t n = context->shape()[0];

  int blocks = std::min(MAX_BLOCKS, (int)n);
  int threads = std::min(MAX_THREADS, (int)k);

  gAttBack<<<blocks, threads>>>(gVa->data(),
                                gContext->data(),
                                gState->data(),
                                gCoverage ? gCoverage->data() : nullptr,

                                va->data(),
                                context->data(),
                                state->data(),
                                coverage ? coverage->data() : nullptr,

                                adj->data(),
                                m,
                                k,
                                n);
}

__global__ void gLNormalization(float* out,
                                const float* in,
                                const float* alpha,
                                const float* beta,
                                int rows,
                                int cols,
                                float eps = 1e-9) {
  extern __shared__ float _share[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0f;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = _sum[0] / cols;
      __syncthreads();

      float* _sqSum = _share + blockDim.x;

      _sqSum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = sp[id] - mean;
          _sqSum[threadIdx.x] += ex * ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sqSum[threadIdx.x] += _sqSum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (_sqSum[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float t = alpha[id] * ((sp[id] - mean) / sigma);
          if(beta != nullptr)
            t += beta[id];
          so[id] = t;
        }
      }
    }
  }
}

void LayerNormalization(
    Tensor out, Tensor in, Tensor gamma, Tensor beta, float eps) {
  hipSetDevice(out->getDevice());

  int rows = in->shape()[0] * in->shape()[2] * in->shape()[3];
  int cols = in->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int)rows);
  int threads = std::min(MAX_THREADS, (int)cols);
  int shared = 2 * threads * sizeof(float);

  gLNormalization<<<blocks, threads, shared>>>(out->data(),
                                               in->data(),
                                               gamma->data(),
                                               beta ? beta->data() : nullptr,
                                               rows,
                                               cols,
                                               eps);
}

__global__ void gLayerNormalizationGrad(float* gradX,
                                        float* gradGamma,
                                        float* gradBeta,
                                        float* adj,
                                        float* y,
                                        float* x,
                                        float* gamma,
                                        float* beta,
                                        int rows,
                                        int cols,
                                        float eps = 1e-9) {
  extern __shared__ float shared[];

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* sum_adj = shared;
      float* sum_adj_x = shared + blockDim.x;
      float* sum_x = shared + 2 * blockDim.x;
      float* sum_sqr = shared + 3 * blockDim.x;

      const float* xRow = x + j * cols;
      const float* yRow = y + j * cols;
      const float* adjRow = adj + j * cols;
      float* gradXRow = gradX + j * cols;

      sum_x[threadIdx.x] = 0.0f;
      sum_adj[threadIdx.x] = 0.0f;
      sum_adj_x[threadIdx.x] = 0.0f;
      sum_sqr[threadIdx.x] = 0.0f;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          sum_x[threadIdx.x] += xRow[id];
          sum_adj_x[threadIdx.x]
              += adjRow[id] * (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          sum_adj[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          sum_x[threadIdx.x] += sum_x[threadIdx.x + skip];
          sum_adj[threadIdx.x] += sum_adj[threadIdx.x + skip];
          sum_adj_x[threadIdx.x] += sum_adj_x[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float mean = sum_x[0] / cols;
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = xRow[id] - mean;
          sum_sqr[threadIdx.x] += ex * ex;
        }
      }

      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          sum_sqr[threadIdx.x] += sum_sqr[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float sigma = sqrtf(eps + (sum_sqr[0] / cols));
      __syncthreads();

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float grad_x = 0.0f;
          float x_hat = (yRow[id] - ((beta) ? beta[id] : 0)) / gamma[id];
          grad_x += cols * adjRow[id];
          grad_x -= sum_adj[0];
          grad_x -= sum_adj_x[0] * x_hat;
          grad_x /= (cols * sigma);

          gradXRow[id] += gamma[id] * grad_x;
          atomicAdd(gradGamma + id, adjRow[id] * x_hat);
          if(beta) {
            atomicAdd(gradBeta + id, adjRow[id]);
          }
        }
      }
    }
  }
}

void LayerNormalizationGrad(Tensor gradX,
                            Tensor gradGamma,
                            Tensor gradBeta,
                            Tensor adj,
                            Tensor y,
                            Tensor x,
                            Tensor gamma,
                            Tensor beta,
                            float eps) {
  hipSetDevice(adj->getDevice());
  int rows = y->shape()[0] * y->shape()[2] * y->shape()[3];
  int cols = y->shape()[1];

  int threads = std::min(MAX_THREADS, cols);
  int blocks = std::min(MAX_BLOCKS, rows);
  int shared = sizeof(float) * threads * 4;

  gLayerNormalizationGrad<<<blocks, threads, shared>>>(
      gradX->data(),
      gradGamma->data(),
      (gradBeta) ? gradBeta->data() : nullptr,
      adj->data(),
      y->data(),
      x->data(),
      gamma->data(),
      (beta) ? beta->data() : nullptr,
      rows,
      cols, eps);
}

__global__ void gShift(float* out, const float* in, int length, int offset) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      if(index - offset < 0 || index - offset >= length)
        out[index] = 0;
      else
        out[index] = in[index - offset];
    }
  }
}

void Shift(Tensor out, Tensor in, ShapeGPU shift, bool invert) {
  int offset
      = in->shape().stride(0) * shift[0] + in->shape().stride(1) * shift[1]
        + in->shape().stride(2) * shift[2] + in->shape().stride(3) * shift[3];

  if(invert)
    offset = -offset;

  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gShift<<<blocks, threads>>>(out->data(), in->data(), length, offset);
}

__global__ void gSetSparse(float* out,
                           const size_t* indices,
                           const float* values,
                           int length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      out[indices[index]] = values[index];
    }
  }
}

void SetSparse(float* out,
               const std::vector<size_t>& indices,
               const std::vector<float>& values) {
  int length = indices.size();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  size_t* d_indices;
  CUDA_CHECK(hipMalloc(&d_indices, length * sizeof(size_t)));
  CUDA_CHECK(hipMemcpy(d_indices,
                        indices.data(),
                        length * sizeof(size_t),
                        hipMemcpyHostToDevice));

  float* d_values;
  CUDA_CHECK(hipMalloc(&d_values, length * sizeof(float)));
  CUDA_CHECK(hipMemcpy(
      d_values, values.data(), length * sizeof(float), hipMemcpyHostToDevice));

  gSetSparse<<<blocks, threads>>>(out, d_indices, d_values, length);

  hipFree(d_indices);
  hipFree(d_values);
}

/******************************************************************************/

__device__ inline float logit(float x) {
  return 1.0f / (1.0f + expf(-x));
}

__global__ void gLSTMCellForward(float* out,
                                 const float* cell,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 size_t rows,
                                 size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float gf = logit(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = logit(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float cout = gf * rowCell[i] + gi * gc;
          rowOut[i] = m * cout + (1 - m) * rowCell[i];
        }
      }
    }
  }
}

void LSTMCellForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape()[0] * out->shape()[2] * out->shape()[3];
  int cols = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // cell state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      rows,
      cols);
}

__global__ void gLSTMOutputForward(float* out,
                                 const float* cell,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 size_t rows,
                                 size_t cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols;
      const float* rowCell = cell + j * cols;

      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {

          int k = i + 3 * cols ;
          float go = logit(xWrow[k] + sUrow[k] + b[k]);

          rowOut[i] = go * tanhf(rowCell[i]);
        }
      }
    }
  }
}

void LSTMOutputForward(Tensor out, std::vector<Tensor> inputs) {
  hipSetDevice(out->getDevice());

  int rows = out->shape()[0] * out->shape()[2] * out->shape()[3];
  int cols = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputForward<<<blocks, threads>>>(
      out->data(),                                // output
      inputs[0]->data(),                          // cell state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      rows,
      cols);
}

__global__ void gLSTMCellBackward(float* outCell,
                                  float* outXW,
                                  float* outSU,
                                  float* outB,
                                  const float* cell,
                                  const float* xW,
                                  const float* sU,
                                  const float* b,
                                  const float* mask,
                                  const float* adj,
                                  size_t rows,
                                  size_t cols) {

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {

          float gf = logit(xWrow[i] + sUrow[i] + b[i]);

          int k = i + cols;
          float gi = logit(xWrow[k] + sUrow[k] + b[k]);

          int l = i + 2 * cols;
          float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += (m * gf - m + 1) * adj;

          // dc/d(b_f) = dc/d(xW_f) ...
          float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
          if(outXW)
            rowOutXW[i] += dcdxf;
          if(outSU)
            rowOutSU[i] += dcdxf;
          if(outB)
            atomicAdd(outB + i, dcdxf);

          // dc/d(b_i) ...
          float dcdb_i = m * gc * gi * (1 - gi) * adj;
          if(outXW)
            rowOutXW[k] += dcdb_i;
          if(outSU)
            rowOutSU[k] += dcdb_i;
          if(outB)
            atomicAdd(outB + k, dcdb_i);

          // dc/d(b_c) ...
          float dcdxc = m * gi * (1 - gc * gc) * adj;
          if(outXW)
            rowOutXW[l] += dcdxc;
          if(outSU)
            rowOutSU[l] += dcdxc;
          if(outB)
            atomicAdd(outB + l, dcdxc);
        }
      }
    }
  }
}

void LSTMCellBackward(std::vector<Tensor> outputs,
                      std::vector<Tensor> inputs,
                      Tensor adj) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape()[0] * adj->shape()[2] * adj->shape()[3];
  int cols = adj->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMCellBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      inputs.size() > 4 ? inputs[4]->data() : 0,  // mask
      adj->data(),
      rows,
      cols);
}

__global__ void gLSTMOutputBackward(float* outCell,
                                  float* outXW,
                                  float* outSU,
                                  float* outB,
                                  const float* cell,
                                  const float* xW,
                                  const float* sU,
                                  const float* b,
                                  const float* adj,
                                  size_t rows,
                                  size_t cols) {

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOutCell = outCell + j * cols;
      float* rowOutXW = outXW + j * cols * 4;
      float* rowOutSU = outSU + j * cols * 4;

      const float* rowCell = cell + j * cols;
      const float* xWrow = xW + j * cols * 4;
      const float* sUrow = sU + j * cols * 4;

      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {

          int k = i + 3 * cols;
          float go = logit(xWrow[k] + sUrow[k] + b[k]);

          float t = tanhf(rowCell[i]);

          float adj = rowAdj[i];

          // dc/dc_{t-1}
          if(outCell)
            rowOutCell[i] += go * (1 - t * t) * adj;

          // dc/d(b_o) = dc/d(xW_f) ...
          float dcdxo = t * go * (1 - go) * adj;
          if(outXW)
            rowOutXW[k] += dcdxo;
          if(outSU)
            rowOutSU[k] += dcdxo;
          if(outB)
            atomicAdd(outB + k, dcdxo);

        }
      }
    }
  }
}

void LSTMOutputBackward(std::vector<Tensor> outputs,
                      std::vector<Tensor> inputs,
                      Tensor adj) {
  hipSetDevice(adj->getDevice());

  int rows = adj->shape()[0] * adj->shape()[2] * adj->shape()[3];
  int cols = adj->shape()[1];

  int blocks = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gLSTMOutputBackward<<<blocks, threads>>>(
      outputs[0] ? outputs[0]->data() : 0,        // state - adj
      outputs[1] ? outputs[1]->data() : 0,        // xW - adj
      outputs[2] ? outputs[2]->data() : 0,        // sU - adj
      outputs[3] ? outputs[3]->data() : 0,        // b - adj
      inputs[0]->data(),                          // state
      inputs[1]->data(),                          // xW
      inputs[2]->data(),                          // sU
      inputs[3]->data(),                          // b
      adj->data(),
      rows,
      cols);
}

__global__ void gHighwayForward(float* out,
                                const float* in1,
                                const float* in2,
                                const float* t,
                                size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = 1.f / (1.f + expf(-t[index]));
      out[index] = in1[index] * sigma + in2[index] * (1.f - sigma);
    }
  }
}


void HighwayForward(Tensor out,
                    const Tensor in1, const Tensor in2, const Tensor t) {
  hipSetDevice(out->getDevice());

  int length = out->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayForward<<<blocks, threads>>>(out->data(),
                                       in1->data(),
                                       in2->data(),
                                       t->data(),
                                       length);
}

__global__ void gHighwayBackward(float* out1,
                                 float* out2,
                                 float* outt,
                                 const float* in1,
                                 const float* in2,
                                 const float* t,
                                 const float* adj,
                                 size_t length) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      float sigma = 1.f / (1.f + expf(-t[index]));
      out1[index] = sigma * adj[index];
      out2[index] = (1.f - sigma) * adj[index];
      outt[index] = sigma * (1.f - sigma) * (in1[index] - in2[index]) * adj[index];
    }
  }
}

void HighwayBackward(Tensor out1, Tensor out2, Tensor outt,
                     const Tensor in1, const Tensor in2, const Tensor t,
                     const Tensor adj) {
  hipSetDevice(out1->getDevice());

  int length = out1->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gHighwayBackward<<<blocks, threads>>>(out1->data(),
                                        out2->data(),
                                        outt->data(),
                                        in1->data(),
                                        in2->data(),
                                        t->data(),
                                        adj->data(),
                                        length);
}


}  // namespace marian
