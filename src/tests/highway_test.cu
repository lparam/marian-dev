#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

#include "marian.h"
#include "layers/highway.h"

using namespace marian;

bool test_vectors(const std::vector<float>& output, const std::vector<float>& corrent) {
  if (output.size() != corrent.size()) {
    return false;
  }

  for (size_t i = 0; i < output.size(); ++i) {
    if (fabsf(output[i] - corrent[i]) > 0.0001f) {
      return false;
    }
  }
  return true;
}

int main(int argc, char** argv) {
  auto config = Config(argc, argv, false, true);
  auto graph = New<ExpressionGraph>(false);
  graph->setDevice(0);
  graph->reserveWorkspaceMB(128);

  int dimBatch = 2;
  int dimWord = 4;
  int batchLength = 5;
  int numLayers = 1;

  int elemNum = dimBatch * dimWord * batchLength * numLayers;

  std::vector<float> embData(elemNum);
  std::vector<float> embMask(elemNum);

  for (size_t i = 0; i < embData.size(); ++i) {
    embData[i] =  1 / (float(i) + 1);
    if (i < dimBatch * batchLength) {
      embMask[i] = 1;
    }
  }

  auto x = graph->param("x", {dimBatch, dimWord, batchLength},
                        keywords::init=inits::from_vector(embData));

  auto output = Highway("highway", 4)(x);

  debug(x, "X");
  debug(output, "wghiway");

  graph->forward();
  graph->backward();
  return 0;
}
