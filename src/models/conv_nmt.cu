#include "hip/hip_runtime.h"
#include "models/conv_nmt.h"
#include "models/encdec.h"


namespace marian {


ConvEncoderState::ConvEncoderState(Expr aContext, Expr cContext, Expr mask)
  : EncoderStateS2S(aContext, mask),
    convContext_(cContext)
{}


Expr ConvEncoderState::getConvContext() {
  return convContext_;
}


Ptr<EncoderState>
PoolingEncoder::build(Ptr<ExpressionGraph> graph,
                      Ptr<data::CorpusBatch> batch,
                      size_t batchIdx) {
  using namespace keywords;

  int dimSrcVoc = options_->get<std::vector<int>>("dim-vocabs")[batchIdx];
  int dimSrcEmb = options_->get<int>("dim-emb");
  int maxSrcLength = options_->get<int>("max-length");

  float dropoutSrc = inference_ ? 0 : options_->get<float>("dropout-src");

  auto xEmb = Embedding("Wemb", dimSrcVoc, dimSrcEmb)(graph);
  auto posEmb = Embedding("Wemb_pos", maxSrcLength, dimSrcEmb)(graph);

  Expr x, xMask;

  std::tie(x, xMask) = prepareSource(xEmb, posEmb, batch, batchIdx);

  std::string convType = options_->get<std::string>("conv-enc-type");

  if (dropoutSrc) {
    int srcWords = x->shape()[2];
    auto srcWordDrop = graph->dropout(dropoutSrc, {1, 1, srcWords});
    x = dropout(x, mask=srcWordDrop);
  }

  Expr attContext;
  Expr srcContext;
  if (convType == "pooling") {
    attContext = Pooling("enc_pooling")(x, xMask);
    srcContext = x;
  } else if (convType == "avg") {
    attContext = Pooling("enc_pooling")(x, xMask);
    srcContext = x;
  } else if (convType == "full") {
    attContext = Pooling("enc_pooling")(x, xMask);
    srcContext = x;
  }
  /* debug(attContext, "ATT"); */
  /* debug(srcContext, "SRC"); */


  return New<ConvEncoderState>(attContext, srcContext, xMask);
}


std::tuple<Expr, Expr>
PoolingEncoder::prepareSource(Expr emb, Expr posEmb, Ptr<data::CorpusBatch> batch, size_t index) {
  using namespace keywords;


  auto& wordIndeces = batch->at(index)->indeces();

  auto& mask = batch->at(index)->mask();

  std::vector<size_t> posIndeces;

  for (size_t iPos = 0; iPos < batch->at(index)->batchWidth(); ++iPos) {
    for (size_t i = 0; i < batch->at(index)->batchSize(); ++i) {
      if (iPos < posEmb->shape()[0]) {
        posIndeces.push_back(iPos);
      } else {
        posIndeces.push_back(posEmb->shape()[0] - 1);
      }
    }
  }

  int batchSize = batch->size();
  int dimEmb = emb->shape()[1];
  int batchLength = batch->at(index)->batchWidth();

  auto graph = emb->graph();

  auto xWord = reshape(rows(emb, wordIndeces), {batchSize, dimEmb, batchLength});
  auto xPos = reshape(rows(posEmb, posIndeces), {batchSize, dimEmb, batchLength});
  auto x = xWord + xPos;
  auto xMask = graph->constant(shape={batchSize, 1, batchLength}, init=inits::from_vector(mask));
  return std::make_tuple(x, xMask);
}


Ptr<EncoderState>
ConvolutionalEncoder::build(Ptr<ExpressionGraph> graph,
                            Ptr<data::CorpusBatch> batch,
                            size_t batchIdx) {
  using namespace keywords;

  int dimSrcVoc = options_->get<std::vector<int>>("dim-vocabs")[batchIdx];
  int dimSrcEmb = options_->get<int>("dim-emb");
  int maxSrcLength = options_->get<int>("max-length");
  dimSrcVoc += maxSrcLength;

  // bool layerNorm = options_->get<bool>("normalize");

  float dropoutSrc = inference_ ? 0 : options_->get<float>("dropout-src");

  auto xEmb = Embedding("Wemb", dimSrcVoc, dimSrcEmb)(graph);

  Expr x, xMask;
  std::tie(x, xMask) = prepareSource(xEmb, batch, batchIdx);

  if(dropoutSrc) {
    int srcWords = x->shape()[2];
    auto srcWordDrop = graph->dropout(dropoutSrc, {1, 1, srcWords});
    x = dropout(x, mask=srcWordDrop);
  }

  int stackDim = 3;
  auto aContext = Convolution("enc_cnn-a_", 3, 1, 2 * stackDim)(x, xMask);
  auto cContext = Convolution("enc_cnn-c_", 3, 1, stackDim)(x, xMask);

  return New<ConvEncoderState>(aContext, cContext, xMask);
}

std::tuple<Expr, Expr>
ConvolutionalEncoder::prepareSource(Expr emb, Ptr<data::CorpusBatch> batch, size_t index) {
  using namespace keywords;

  auto& wordIndeces = batch->at(index)->indeces();
  auto& mask = batch->at(index)->mask();
  std::vector<size_t> posIndeces;

  int dimSrcVoc = options_->get<std::vector<int>>("dim-vocabs")[index];

  for (size_t iPos = 0; iPos < batch->at(index)->batchSize(); ++iPos) {
    posIndeces.push_back(dimSrcVoc + iPos);
  }

  int dimBatch = batch->size();
  int dimEmb = emb->shape()[1];
  int dimWords = batch->at(index)->batchSize();

  auto graph = emb->graph();

  auto xWord = reshape(rows(emb, wordIndeces), {dimBatch, dimEmb, dimWords});
  auto xPos = reshape(rows(emb, posIndeces), {dimBatch, dimEmb, dimWords});
  auto x = xWord + xPos;
  auto xMask = graph->constant(shape={dimBatch, 1, dimWords},
                                init=inits::from_vector(mask));
  return std::make_tuple(x, xMask);
}

Ptr<DecoderState> ConvolutionalDecoder::startState(Ptr<EncoderState> encState) {
  using namespace keywords;

  auto meanContext =
    weighted_average(std::static_pointer_cast<ConvEncoderState>(encState)->getConvContext(),
                     encState->getMask(),
                     axis=2);

  bool layerNorm = options_->get<bool>("normalize");
  auto start = Dense("ff_state",
                     options_->get<int>("dim-rnn"),
                     activation=act::tanh,
                     normalize=layerNorm)(meanContext);
  std::vector<Expr> startStates(options_->get<size_t>("layers-dec"), start);
  return New<DecoderStateS2S>(startStates, nullptr, encState);
}

Ptr<DecoderState>
ConvolutionalDecoder::step(Expr embeddings, Ptr<DecoderState> state, bool single) {
  using namespace keywords;

  int dimTrgVoc = options_->get<std::vector<int>>("dim-vocabs").back();
  int dimTrgEmb = options_->get<int>("dim-emb");
  int dimDecState = options_->get<int>("dim-rnn");
  bool layerNorm = options_->get<bool>("normalize");
  bool skipDepth = options_->get<bool>("skip");
  size_t decoderLayers = options_->get<size_t>("layers-dec");

  float dropoutRnn = inference_ ? 0 : options_->get<float>("dropout-rnn");
  float dropoutTrg = inference_ ? 0 : options_->get<float>("dropout-trg");

  auto graph = embeddings->graph();

  if(dropoutTrg) {
    int trgWords = embeddings->shape()[2];
    auto trgWordDrop = graph->dropout(dropoutTrg, {1, 1, trgWords});
    embeddings = dropout(embeddings, mask=trgWordDrop);
  }

  if (!attention_)
    attention_ = New<GlobalAttention>("decoder",
                                      state->getEncoderState(),
                                      std::static_pointer_cast<ConvEncoderState>(state->getEncoderState())->getConvContext(),
                                      dimDecState,
                                      dropout_prob=dropoutRnn,
                                      normalize=layerNorm);
  RNN<CGRU> rnnL1(graph, "decoder",
                  dimTrgEmb, dimDecState,
                  attention_,
                  dropout_prob=dropoutRnn,
                  normalize=layerNorm);

  auto stateS2S = std::dynamic_pointer_cast<DecoderStateS2S>(state);
  auto stateL1 = rnnL1(embeddings, stateS2S->getStates()[0]);
  auto alignedContext = single ?
    rnnL1.getCell()->getLastContext() :
    rnnL1.getCell()->getContexts();

  std::vector<Expr> statesOut;
  statesOut.push_back(stateL1);

  Expr outputLn;
  if(decoderLayers > 1) {
    std::vector<Expr> statesIn;
    for(int i = 1; i < stateS2S->getStates().size(); ++i)
      statesIn.push_back(stateS2S->getStates()[i]);

    std::vector<Expr> statesLn;
    std::tie(outputLn, statesLn) = MLRNN<GRU>(graph, "decoder",
                                              decoderLayers - 1,
                                              dimDecState, dimDecState,
                                              normalize=layerNorm,
                                              dropout_prob=dropoutRnn,
                                              skip=skipDepth,
                                              skip_first=skipDepth)
                                              (stateL1, statesIn);

    statesOut.insert(statesOut.end(),
                      statesLn.begin(), statesLn.end());
  }
  else {
    outputLn = stateL1;
  }

  //// 2-layer feedforward network for outputs and cost
  auto logitsL1 = Dense("ff_logit_l1", dimTrgEmb,
                        activation=act::tanh,
                        normalize=layerNorm)
                    (embeddings, outputLn, alignedContext);

  auto logitsOut = filterInfo_ ?
    DenseWithFilter("ff_logit_l2", dimTrgVoc, filterInfo_->indeces())(logitsL1) :
    Dense("ff_logit_l2", dimTrgVoc)(logitsL1);

  return New<DecoderStateS2S>(statesOut, logitsOut,
                              state->getEncoderState());
}


}  // namespace marian
