#include <algorithm>
#include <chrono>
#include <cstdio>
#include <iomanip>
#include <string>

#include <boost/timer/timer.hpp>

#include "marian.h"

#include "examples/mnist/model.h"
#include "examples/mnist/training.h"
#include "training/graph_group.h"

const std::vector<std::string> TRAIN_SET
    = {"../src/examples/mnist/train-images-idx3-ubyte",
       "../src/examples/mnist/train-labels-idx1-ubyte"};
const std::vector<std::string> VALID_SET
    = {"../src/examples/mnist/t10k-images-idx3-ubyte",
       "../src/examples/mnist/t10k-labels-idx1-ubyte"};

using namespace marian;

int main(int argc, char** argv) {
  auto options = New<Config>(argc, argv, ConfigMode::training, false);

  if(!options->has("train-sets"))
    options->set("train-sets", TRAIN_SET);
  if(!options->has("valid-sets"))
    options->set("valid-sets", VALID_SET);

  auto devices = options->get<std::vector<size_t>>("devices");

  if(devices.size() > 1)
    New<TrainMNIST<AsyncGraphGroup<models::MNISTModel>>>(options)->run();
  else
    New<TrainMNIST<SingletonGraph<models::MNISTModel>>>(options)->run();

  return 0;
}
