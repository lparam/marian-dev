#include "hip/hip_runtime.h"
#include "graph/expression_operators.h"
#include "graph/node_operators.h"
#include "graph/node_operators_unary.h"

namespace marian {

Expr debug(Expr a, const std::string& message) {
  a->debug(message);
  return a;
}

Expr rows(Expr a, const std::vector<size_t>& indeces) {
  return Expression<RowsNodeOp>(a, indeces);
}

Expr cols(Expr a, const std::vector<size_t>& indeces) {
  return Expression<ColsNodeOp>(a, indeces);
}

Expr logit(Expr a) {
  return Expression<LogitNodeOp>(a);
}

Expr relu(Expr a) {
  return Expression<ReLUNodeOp>(a);
}

Expr log(Expr a) {
  return Expression<LogNodeOp>(a);
};

Expr exp(Expr a) {
  return Expression<ExpNodeOp>(a);
};

Expr operator-(Expr a) {
  return Expression<NegNodeOp>(a);
};

Expr softmax(Expr a, Expr mask) {
  return Expression<SoftmaxNodeOp>(a, mask);
}

Expr logsoftmax(Expr a) {
  return Expression<LogSoftmaxNodeOp>(a);
}

/*********************************************************/

Expr operator+(Expr a, Expr b) {
  return Expression<PlusNodeOp>(a, b);
}

Expr operator-(Expr a, Expr b) {
  return Expression<MinusNodeOp>(a, b);
}

Expr operator*(Expr a, Expr b) {
  return Expression<MultNodeOp>(a, b);
}

Expr operator/(Expr a, Expr b) {
  return Expression<DivNodeOp>(a, b);
}

/*********************************************************/

Expr operator+(Expr a, float b) {
  return Expression<ScalarAddNodeOp>(a, b);
}

Expr operator+(float a, Expr b) {
  return Expression<ScalarAddNodeOp>(b, a);
}

Expr operator-(Expr a, float b) {
  return Expression<ScalarAddNodeOp>(a, -b);
}

Expr operator-(float a, Expr b) {
  return Expression<ScalarAddNodeOp>(-b, a);
}

Expr operator*(float a, Expr b) {
  return Expression<ScalarMultNodeOp>(b, a);  
}

Expr operator*(Expr a, float b) {
  return Expression<ScalarMultNodeOp>(a, b);  
}

Expr operator/(Expr a, float b) {
  return Expression<ScalarMultNodeOp>(a, 1.f / b);  
}

/*********************************************************/

Expr dot(Expr a, Expr b) {
  return Expression<DotNodeOp>(a, b);
}

Expr transpose(Expr a) {
  return Expression<TransposeNodeOp>(a);
}

Expr step(Expr a, size_t step) {
  return Expression<TimestepNodeOp>(a, step);
}

Expr cross_entropy(Expr a, Expr b) {
  auto sOrig = a->shape();
  auto sOut = a->shape();
  Shape sTemp({sOrig[0] * sOrig[2] * sOrig[3], sOrig[1], 1, 1});
  sOut.set(1, 1);
  return reshape(Expression<CrossEntropyNodeOp>(reshape(a, sTemp), b), sOut);
}

Expr affine(Expr a, Expr b, Expr c) {
  std::vector<Expr> nodes = {a, b, c};
  return Expression<AffineNodeOp>(nodes);
}

Expr plus(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr tanh(const std::vector<Expr>& nodes) {
  return Expression<TanhNodeOp>(nodes);
}

Expr logit(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr relu(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr sqrt(Expr a, float eps) {
  return Expression<SqrtNodeOp>(a, eps);
}

Expr square(Expr a) {
  return Expression<SquareNodeOp>(a);
}

Expr layer_norm(Expr x, Expr gamma, Expr beta) {
  std::vector<Expr> nodes = {x, gamma};
  if(beta)
    nodes.push_back(beta);
  return Expression<LayerNormalizationOp>(nodes);
}

Expr shift(Expr a, Shape shift) {
  return Expression<ShiftNodeOp>(a, shift);
}

Expr lexical_bias(Expr logits, Expr att, float eps, Ptr<sparse::CSR> lf) {
  return Expression<LexicalProbNodeOp>(logits, att, eps, lf);
}

#ifdef CUDNN

Expr convolution(Expr x, Expr filters) {
  std::vector<Expr> nodes = {x, filters};
  return Expression<ConvolutionOp>(nodes);
}

Expr avg_pooling(Expr x) {
  return Expression<MaxPoolingOp>(x);
}

#endif

}
