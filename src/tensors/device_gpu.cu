#include "tensors/device_gpu.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include "kernels/cuda_helpers.h"

namespace marian {

DeviceGPU::~DeviceGPU() {
  hipSetDevice(device_);
  if(data_) {
    CUDA_CHECK(hipFree(data_));
  }
  hipDeviceSynchronize();
}

void DeviceGPU::reserve(size_t size) {
  size = align(size);
  hipSetDevice(device_);

  UTIL_THROW_IF2(size < size_, "New size must be larger than old size");

  if(data_) {
    // Allocate memory by going through host memory
    uint8_t *temp = new uint8_t[size_];
    CUDA_CHECK(hipMemcpy(temp, data_, size_, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(data_));
    CUDA_CHECK(hipMalloc(&data_, size));
    CUDA_CHECK(hipMemcpy(data_, temp, size_, hipMemcpyHostToDevice));
    delete[] temp;
  } else {
    CUDA_CHECK(hipMalloc(&data_, size));
  }

  size_ = size;
}

}